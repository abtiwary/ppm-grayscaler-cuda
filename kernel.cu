#include "hip/hip_runtime.h"

#include <cstdint>
#include <iostream>

#include "kernel.cuh"
#include "hip/hip_runtime.h"

__global__
void grayscale_gpu(uint8_t* in_image, uint8_t* out_image, int width, int height) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    int index = (y * width + x) * 3;

    if (x < width && y < height) {
        uint8_t r = in_image[index + 0];
        uint8_t g = in_image[index + 1];
        uint8_t b = in_image[index + 2];

        uint8_t val = (uint8_t)(0.30*(float)r + 0.59*(float)g + 0.11*(float)b);
         
        out_image[index + 0] = val;
        out_image[index + 1] = val;
        out_image[index + 2] = val;
    }
    
}

void grayscaler(int width, int height, uint8_t* image_data) {
    uint8_t* host_img = (uint8_t*)malloc(width * height * 3);
    if (!host_img) {
        fprintf(stderr, "could not malloc the host image!\n");
        return;
    }

    uint8_t* device_in_img;
    hipMalloc(&device_in_img, width * height * 3);
    hipMemcpy(device_in_img, image_data, width * height * 3, hipMemcpyHostToDevice);

    uint8_t* device_out_img;
    hipMalloc(&device_out_img, width * height * 3);

    dim3 blocks(32, 32);
    dim3 grids(ceil((float)width / 32), ceil((float)height / 32));
    
    grayscale_gpu<<<grids, blocks>>>(device_in_img, device_out_img, width, height);
    hipDeviceSynchronize();
    
    hipMemcpy(host_img, device_out_img, width * height * 3, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // write an output image
    FILE* outfile = fopen("/home/pimeson/temp/grayscale_cuda.ppm", "w+");
    fprintf(outfile, "P6\n%d %d 255\n", width, height);
    fwrite(host_img, width * height * 3, sizeof(uint8_t), outfile); 
    fclose(outfile);
    
    free(host_img);
    hipFree(device_in_img);
    hipFree(device_out_img);

    std::cout << "wrote an image\n";
}

